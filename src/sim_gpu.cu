#include <math.h>
#include <iomanip>
#include <iostream>
#include <cstring>
#include <chrono>
#include "hip/hip_runtime.h"


using std::endl;
using std::cout;

#define THREADS 512

inline constexpr double R_1  = 0.501306994212753;

__global__ void simChunk(double* y_min, double* y_max, long int* r1_wins, long int* r2_wins, long int* trials, double delta) {
    int i = threadIdx.x;
    for (double x = 0.0; x <= 1.0; x += delta) {
        for (double y = y_min[i]; y <= y_max[i]; y += delta) {
            double r_target = sqrt(x*x + y*y);
            // Outside of game area
            if (r_target > 1.0) continue;
            trials[i]++;
            // Robot 2 wins by default
            if (r_target <= R_1/2) {
                r2_wins[i]++;
                continue;
            }
            double r1_distance = abs(r_target - R_1);
            // Robot 2's optimal strategy given the target's radius
            double r2_x = sqrt((2*r_target*R_1) - (R_1*R_1));
            double r2_distance = sqrt((x - r2_x)*(x - r2_x) + y*y);
            if (r2_distance <= r1_distance) {
                r2_wins[i]++;
            } else {
                r1_wins[i]++;
            }
        }   
    }
}

int main(int argc, char* argv[]) {
    int accuracy = 10;
    for (int i = 1; i < argc; i++) {
        if (std::strcmp(argv[i], "-a") == 0) {
            if (i + 1 == argc) {
                cout << "Accuracy not given: defaulting to 10" << endl;
            } else {
                try {
                    accuracy = std::stoi(argv[i+1]);
                    if (accuracy < 1) {
                        cout << "Invalid accuracy: defaulting to 10" << endl;
                        accuracy = 10;
                    }
                } catch (std::exception &err) {
                    cout << "Invalid accuracy: defaulting to 10" << endl;
                    accuracy = 10;
                }
            }
        }
    }
    auto start = std::chrono::high_resolution_clock::now();
    double delta = pow(0.5, accuracy);

    double y_min[THREADS] = {0};
    double y_max[THREADS] = {0};
    long int r1_wins[THREADS] = {0};
    long int r2_wins[THREADS] = {0};
    long int trials[THREADS] = {0};

    for (int i = 0; i < THREADS; i++) {
        y_min[i] = (1.0/THREADS)*(double)i;
        y_max[i] = (1.0/THREADS)*(double)(i+1) - delta;
    }
    y_max[THREADS] = 1.0;
    
    double* cuda_y_min = 0;
    double* cuda_y_max = 0;
    long int* cuda_r1_wins = 0;
    long int* cuda_r2_wins = 0;
    long int* cuda_trials = 0;

    hipMalloc(&cuda_y_min, sizeof(y_min));
    hipMalloc(&cuda_y_max, sizeof(y_max));
    hipMalloc(&cuda_r1_wins, sizeof(r1_wins));
    hipMalloc(&cuda_r2_wins, sizeof(r2_wins));
    hipMalloc(&cuda_trials, sizeof(trials));

    hipMemcpy(cuda_y_min, y_min, sizeof(y_min), hipMemcpyHostToDevice);
    hipMemcpy(cuda_y_max, y_max, sizeof(y_max), hipMemcpyHostToDevice);

    simChunk <<< 1, THREADS >>> (cuda_y_min, cuda_y_max, cuda_r1_wins, cuda_r2_wins, cuda_trials, delta);

    hipMemcpy(r1_wins, cuda_r1_wins, sizeof(r1_wins), hipMemcpyDeviceToHost);
    hipMemcpy(r2_wins, cuda_r2_wins, sizeof(r2_wins), hipMemcpyDeviceToHost);
    hipMemcpy(trials, cuda_trials, sizeof(trials), hipMemcpyDeviceToHost);

    long int r1_total, r2_total, total_trials = 0;

    for (int i = 0; i < THREADS; i++) {
        r1_total += r1_wins[i];
        r2_total += r2_wins[i];
        total_trials += trials[i];
    }

    double temp = R_1*R_1/8;

    double robot1_winrate = r1_total/(double)(total_trials*2) + 0.5 - temp;
    double robot2_winrate = r2_total/(double)(total_trials*2) + temp;

    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
    
    cout << endl << std::setprecision(10) << "Robot 1 winrate: "<< robot1_winrate << endl << "Robot 2 winrate: " << robot2_winrate << endl;
    cout << "Trials: " << total_trials << endl;
    cout << "Took:   " << duration.count() << "ms" << endl;
    return 0;
}